#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void vectorAddKernel(float *A, float *B, float *C, int N){

  int id = gridDim.x * blockIdx.x + threadIdx.x;
  int stride = gridDim.x * blockDim.x;
  //printf("Entered Kernel\n");
  for(int i=id; i<N; i+=stride){
    //printf("A[%d]:%f, B[%d]:%f\n",i,A[i],i,B[i]);
    C[i] = A[i] + B[i];
  }  
}

// Function to process inputs and return outputs to Python
extern "C" void vectorAdd(float *A, float *B, float *C, int N) {
    // This function is called from Python
    // Input:
    // - A, B: Input vectors
    // - C: Output vector (will be modified in-place)
    // - N: Size of the vectors
  float *A_gpu, *B_gpu, *C_gpu;
  int size = sizeof(int)*N;
  
  hipMalloc(&A_gpu, size);
  hipMalloc(&B_gpu, size);
  hipMalloc(&C_gpu, size);

  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  
  hipEventRecord(start);

  hipMemcpy(A_gpu,  A, size, hipMemcpyHostToDevice);
  hipMemcpy(B_gpu,  B, size, hipMemcpyHostToDevice);
  hipMemcpy(C_gpu,  C, size, hipMemcpyHostToDevice);
  
  vectorAddKernel<<<128,512>>>(A_gpu,B_gpu,C_gpu,N);
  
  hipMemcpy(A, A_gpu, size, hipMemcpyDeviceToHost);
  hipMemcpy(B, B_gpu, size, hipMemcpyDeviceToHost);
  hipMemcpy(C, C_gpu, size, hipMemcpyDeviceToHost);

  hipEventRecord(stop);
  hipEventSynchronize(stop);  
  
  hipDeviceSynchronize();

  hipFree(A_gpu);
  hipFree(B_gpu);
  hipFree(C_gpu);


  float milliseconds = 0;
  hipEventElapsedTime(&milliseconds, start, stop);
  printf("Elapsed time: %f ms\n", milliseconds);

  hipEventDestroy(start);
  hipEventDestroy(stop);
}
